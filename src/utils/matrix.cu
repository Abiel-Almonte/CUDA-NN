#include "matrix.hh"

Matrix::Matrix(size_t x, size_t y):shape(x, y), device_data(nullptr), host_data(nullptr), device_allocated(false), host_allocated(false){}

Matrix::Matrix(Shape shape):Matrix(shape.x, shape.y){}

void Matrix::allocateHost() {

	if (!host_allocated) {
		host_data= shared_ptr<float>(new float[shape.x * shape.y],[&](float* ptr){delete[] ptr;});
		host_allocated= true;
	}

}

void Matrix::allocateCuda() {

	if (!device_allocated) {
		float* temp= nullptr;
		hipError_t err= hipMalloc(&temp, shape.x * shape.y * sizeof(float));

		if (err != hipSuccess) {
			cerr << "hipMalloc failed: " << hipGetErrorString(err) << endl;
			exit(err);
    	}

		device_data = shared_ptr<float>(temp, [&](float* ptr){hipFree(ptr);});
		device_allocated = true;
	}

}


void Matrix::allocateMemory() {
	allocateCuda();
	allocateHost();
}

void Matrix::allocateMemoryIfNotAllocated(Shape shape) {

	if (!device_allocated && !host_allocated) {
		this->shape= shape;
		allocateMemory();
	}

}

void Matrix::copyDeviceToHost() {
	if (device_allocated && host_allocated) {
		hipError_t err=  hipMemcpy(host_data.get(), device_data.get(), shape.x * shape.y * sizeof(float), hipMemcpyDeviceToHost);

		if (err != hipSuccess) {
			cerr << "hipMalloc failed: " << hipGetErrorString(err) << endl;
			exit(err);
    	}
	}
}
void Matrix::copyHostToDevice() {
	if (device_allocated && host_allocated) {
		hipError_t err=  hipMemcpy(device_data.get(), host_data.get(), shape.x * shape.y * sizeof(float), hipMemcpyHostToDevice);

		if (err != hipSuccess) {
			cerr << "hipMalloc failed: " << hipGetErrorString(err) << endl;
			exit(err);
    	}
	}
}


float& Matrix::operator[](const int i) {
	return host_data.get()[i];
}

const float& Matrix::operator[](const int i) const {
	return host_data.get()[i];
}